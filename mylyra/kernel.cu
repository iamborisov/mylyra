#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <stdint.h>

#include <Windows.h>
// MSVC defines this in winsock2.h!?
/*typedef struct timeval {
	long tv_sec;
	long tv_usec;
} timeval;*/

int gettimeofday(struct timeval * tp, struct timezone * tzp)
{
	// Note: some broken versions only have 8 trailing zero's, the correct epoch has 9 trailing zero's
	// This magic number is the number of 100 nanosecond intervals since January 1, 1601 (UTC)
	// until 00:00:00 January 1, 1970 
	static const uint64_t EPOCH = ((uint64_t)116444736000000000ULL);

	SYSTEMTIME  system_time;
	FILETIME    file_time;
	uint64_t    time;

	GetSystemTime(&system_time);
	SystemTimeToFileTime(&system_time, &file_time);
	time = ((uint64_t)file_time.dwLowDateTime);
	time += ((uint64_t)file_time.dwHighDateTime) << 32;

	tp->tv_sec = (long)((time - EPOCH) / 10000000L);
	tp->tv_usec = (long)(system_time.wMilliseconds * 1000);
	return 0;
}


typedef unsigned char byte;

#ifndef N_COLS
#define N_COLS 8
#endif

#ifndef nPARALLEL
#define nPARALLEL 2
#endif

#define ROW_LEN_INT64 (BLOCK_LEN_INT64 * N_COLS)                //Total length of a row: N_COLS blocks
#define ROW_LEN_BYTES (ROW_LEN_INT64 * 8)                       //Number of bytes per row

#if defined(__GNUC__)
#define ALIGN __attribute__ ((aligned(32)))
#elif defined(_MSC_VER)
#define ALIGN __declspec(align(32))
#else
#define ALIGN
#endif

//Block length required so Blake2's Initialization Vector (IV) is not overwritten (THIS SHOULD NOT BE MODIFIED)
#define BLOCK_LEN_BLAKE2_SAFE_INT64 8                                   //512 bits (=64 bytes, =8 uint64_t)
#define BLOCK_LEN_BLAKE2_SAFE_BYTES (BLOCK_LEN_BLAKE2_SAFE_INT64 * 8)   //same as above, in bytes

//default block lenght: 768 bits
#ifndef BLOCK_LEN_INT64
#define BLOCK_LEN_INT64 12                                      //Block length: 768 bits (=96 bytes, =12 uint64_t)
#endif

#define BLOCK_LEN_BYTES (BLOCK_LEN_INT64 * 8)                           //Block length, in bytes

#define STATESIZE_INT64 16
#define STATESIZE_BYTES (16 * sizeof (uint64_t))

#ifndef RHO
#define RHO 1                                                   //Number of reduced rounds performed
#endif

/*Blake2b IV Array*/
__device__ static const uint64_t blake2b_IV[8] =
{
	0x6a09e667f3bcc908ULL, 0xbb67ae8584caa73bULL,
	0x3c6ef372fe94f82bULL, 0xa54ff53a5f1d36f1ULL,
	0x510e527fade682d1ULL, 0x9b05688c2b3e6c1fULL,
	0x1f83d9abfb41bd6bULL, 0x5be0cd19137e2179ULL
};

/*Blake2b's rotation*/
__device__ static inline uint64_t rotr64(const uint64_t w, const unsigned c){
	return (w >> c) | (w << (64 - c));
}

/*Main change compared with Blake2b*/
__device__ static inline uint64_t fBlaMka(uint64_t x, uint64_t y){
	uint32_t lessX = (uint32_t)x;
	uint32_t lessY = (uint32_t)y;

	uint64_t lessZ = (uint64_t)lessX;
	lessZ = lessZ * lessY;
	lessZ = lessZ << 1;

	uint64_t z = lessZ + x + y;

	return z;
}

#define DIAGONALIZE(r,v) \
    t0=v[4];                      v[4]=v[5]; v[5]=v[6]; v[6]=v[7]; v[7]=t0; \
    t0=v[8]; t1=v[9];             v[8]=v[10]; v[9]=v[11]; v[10]=t0; v[11]=t1; \
    t0=v[12]; t1=v[13]; t2=v[14]; v[12]=v[15]; v[13]=t0; v[14]=t1; v[15]=t2;

/*Blake2b's G function*/
#define G(r,i,a,b,c,d) \
  do { \
    a = a + b; \
    d = rotr64(d ^ a, 32); \
    c = c + d; \
    b = rotr64(b ^ c, 24); \
    a = a + b; \
    d = rotr64(d ^ a, 16); \
    c = c + d; \
    b = rotr64(b ^ c, 63); \
    } while(0)

/*One Round of the Blake2b's compression function*/
#define ROUND_LYRA(r)  \
    G(r,0,v[ 0],v[ 4],v[ 8],v[12]); \
    G(r,1,v[ 1],v[ 5],v[ 9],v[13]); \
    G(r,2,v[ 2],v[ 6],v[10],v[14]); \
    G(r,3,v[ 3],v[ 7],v[11],v[15]); \
    G(r,4,v[ 0],v[ 5],v[10],v[15]); \
    G(r,5,v[ 1],v[ 6],v[11],v[12]); \
    G(r,6,v[ 2],v[ 7],v[ 8],v[13]); \
    G(r,7,v[ 3],v[ 4],v[ 9],v[14]);


//---- Initialization
__global__ void bootStrapGPU(uint64_t * memMatrixGPU, unsigned char * pkeysGPU, unsigned int kLen, unsigned char *pwdGPU, unsigned int pwdlen, unsigned char *saltGPU, unsigned int saltlen, unsigned int timeCost, unsigned int nRows, unsigned int nCols, uint64_t nBlocksInput, unsigned int totalPasswords);

//---- Housekeeping
__global__ void initState(uint64_t state[/*16*/], unsigned int totalPasswords);

//---- Squeezes
__global__ void reducedSqueezeRow0(uint64_t* row, uint64_t* state, unsigned int totalPasswords);
__global__ void squeeze(uint64_t *state, byte *out, unsigned int len, unsigned int totalPasswords);

//---- Absorbs
__global__ void absorbInput(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, uint64_t *stateIdxGPU, uint64_t nBlocksInput, unsigned int totalPasswords);

//---- Duplexes
__global__ void reducedDuplexRow1and2(uint64_t *rowIn, uint64_t *state, unsigned int totalPasswords, int first, int second);

//---- Setup and Wandering
__global__ void setupPhaseWanderingGPU(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, uint64_t sizeSlice, unsigned int totalPasswords, unsigned int timeCost);
__global__ void setupPhaseWanderingGPU_P1(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, uint64_t sizeSlice, unsigned int totalPasswords, unsigned int timeCost);

//---- Misc
void printArray(unsigned char *array, unsigned int size, char *name) {
	int i;
	printf("%s: ", name);
	for (i = 0; i < size; i++) {
		printf("%2x|", array[i]);
	}
	printf("\n");
}

int gpuMult(void *K, unsigned int kLen, unsigned char **passwords, unsigned int pwdlen, unsigned char **salts, unsigned int saltlen, unsigned int timeCost, unsigned int nRows, unsigned int nCols, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize) {
	int result = 0;

	//============================= Basic variables ============================//
#if (nPARALLEL > 1)
	int64_t i, j, k; //auxiliary iteration counter
#endif   // nPARALLEL > 1

	hipError_t errorCUDA;
	uint64_t sizeSlice = nRows / nPARALLEL;
	//==========================================================================/
	uint64_t nBlocksInput;

	//Checks kernel geometry configuration
	if ((gridSize * blockSize) != (totalPasswords * nPARALLEL)) {
		printf("Error in thread geometry: (gridSize * blockSize) != (totalPasswords * nPARALLEL).\n");
		return -1;
	}
	//Checks whether or not the salt+password are within the accepted limits
	if (pwdlen + saltlen > ROW_LEN_BYTES) {
		return -1;
	}

	//========== Initializing the Memory Matrix and Keys =============//
	//Allocates the keys
	unsigned char *pKeys = (unsigned char *)malloc(totalPasswords * nPARALLEL * kLen * sizeof(unsigned char));
	if (pKeys == NULL) {
		return -1;
	}

	// GPU memory matrix alloc:
	// Memory matrix: nRows of nCols blocks, each block having BLOCK_LEN_INT64 64-bit words
	uint64_t *memMatrixGPU;
	errorCUDA = hipMalloc((void**)&memMatrixGPU, totalPasswords * nRows * ROW_LEN_BYTES);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Allocates the GPU keys
	unsigned char *pkeysGPU;
	errorCUDA = hipMalloc((void**)&pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof(unsigned char));
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Sponge state: 16 uint64_t, BLOCK_LEN_INT64 words of them for the bitrate (b) and the remainder for the capacity (c)
	uint64_t *stateThreadGPU;
	errorCUDA = hipMalloc((void**)&stateThreadGPU, totalPasswords * nPARALLEL * STATESIZE_BYTES);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	// stateThreadGPU cleanup:
	hipMemset(stateThreadGPU, 0, totalPasswords * nPARALLEL * STATESIZE_BYTES);
	if (hipSuccess != hipGetLastError()) {
		printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
		return -2;
	}

	//Allocates the State Index to be absorbed by each thread.
	uint64_t *stateIdxGPU;
	errorCUDA = hipMalloc((void**)&stateIdxGPU, totalPasswords * nPARALLEL * BLOCK_LEN_BLAKE2_SAFE_BYTES);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Allocates the Password in GPU.
	unsigned char *pwdGPU;
	errorCUDA = hipMalloc((void**)&pwdGPU, totalPasswords * pwdlen);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	// Transfers the password to GPU.
	errorCUDA = hipMemcpy(pwdGPU, passwords[0], totalPasswords * pwdlen, hipMemcpyHostToDevice);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Allocates the Salt in GPU.
	unsigned char *saltGPU;
	errorCUDA = hipMalloc((void**)&saltGPU, totalPasswords * saltlen);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	// Transfers the salt to GPU.
	errorCUDA = hipMemcpy(saltGPU, salts[0], totalPasswords * saltlen, hipMemcpyHostToDevice);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//========================== BootStrapping Phase ==========================//
#if (nPARALLEL == 1)
	nBlocksInput = ((saltlen + pwdlen + 6 * sizeof(int)) / BLOCK_LEN_BLAKE2_SAFE_BYTES) + 1;
#endif  // nPARALLEL == 1

#if (nPARALLEL > 1)
	nBlocksInput = ((saltlen + pwdlen + 8 * sizeof(int)) / BLOCK_LEN_BLAKE2_SAFE_BYTES) + 1;
#endif   // nPARALLEL > 1

	bootStrapGPU << <gridSize, blockSize >> >(memMatrixGPU, pkeysGPU, kLen, pwdGPU, pwdlen, saltGPU, saltlen, timeCost, nRows, nCols, nBlocksInput, totalPasswords);

	// Needs to wait all threads:
	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//============== Initializing the Sponge State =============/
	initState << <gridSize, blockSize >> >(stateThreadGPU, totalPasswords);

	// Wait all threads to verify execution errors.
	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//============= Absorbing the input data with the sponge ===============//
	absorbInput << <gridSize, blockSize >> >(memMatrixGPU, stateThreadGPU, stateIdxGPU, nBlocksInput, totalPasswords);

	// Wait all threads to verify execution errors.
	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//================================ Setup and Wandering Phase =============================//
	//Initializes M[0]
	reducedSqueezeRow0 << <gridSize, blockSize >> >(memMatrixGPU, stateThreadGPU, totalPasswords); //The locally copied password is most likely overwritten here

	// Wait all threads to verify execution errors.
	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Initializes M[1]
	reducedDuplexRow1and2 << <gridSize, blockSize >> >(memMatrixGPU, stateThreadGPU, totalPasswords, 0, 1);

	// Wait all threads to verify execution errors.
	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Initializes M[2]
	reducedDuplexRow1and2 << <gridSize, blockSize >> >(memMatrixGPU, stateThreadGPU, totalPasswords, 1, 2);

	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

#if (nPARALLEL > 1)
	// Runs Setup and Wandering Phase
	setupPhaseWanderingGPU << <gridSize, blockSize >> >(memMatrixGPU, stateThreadGPU, sizeSlice, totalPasswords, timeCost);
#endif //nParallel > 1

	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error after SetupWandering: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	//Squeezes the keys
	squeeze << <gridSize, blockSize >> >(stateThreadGPU, pkeysGPU, kLen, totalPasswords);

	hipDeviceSynchronize();

	errorCUDA = hipGetLastError();
	if (hipSuccess != errorCUDA) {
		printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}

	// Getting the keys back.
	errorCUDA = hipMemcpy(pKeys, pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (hipSuccess != errorCUDA) {
		printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
		printf("Error: %s \n", hipGetErrorString(errorCUDA));
		return -2;
	}


#if (nPARALLEL > 1)
	// XORs all Keys
	for (k = 0; k < totalPasswords; k++) {
		for (i = 1; i < nPARALLEL; i++) {
			for (j = 0; j < kLen; j++) {
				pKeys[k * kLen * nPARALLEL + j] ^= pKeys[k * kLen * nPARALLEL + i * kLen + j];
			}
		}
	}

	//Move the keys to proper place
	for (k = 1; k < totalPasswords; k++) {
		for (j = 0; j < kLen; j++) {
			pKeys[k * kLen + j] = pKeys[k * kLen * nPARALLEL + j];
		}
	}
#endif //nParallel > 1

	// Returns in the correct variable
	memcpy(K, pKeys, totalPasswords * kLen * sizeof(unsigned char));

	//========== Frees the Memory Matrix and Keys =============//
	hipFree(memMatrixGPU);
	hipFree(pkeysGPU);
	hipFree(stateThreadGPU);
	hipFree(stateIdxGPU);
	hipFree(saltGPU);
	hipFree(pwdGPU);

	//Free allKeys
	free(pKeys);
	pKeys = NULL;

	return result;
}


//#if (nPARALLEL > 1)
__device__ uint64_t sizeSlicedRows;
//#endif //nParallel > 1

/**
* Execute G function, with all 12 rounds for Blake2 and  BlaMka, and 24 round for half-round BlaMka.
*
* @param v     A 1024-bit (16 uint64_t) array to be processed by Blake2b's or BlaMka's G function
*/
__device__ inline static void spongeLyra(uint64_t *v) {
	int i;

	for (i = 0; i < 12; i++){
		ROUND_LYRA(i);
	}
}

/**
* Executes a reduced version of G function with only RHO round
* @param v     A 1024-bit (16 uint64_t) array to be processed by Blake2b's or BlaMka's G function
*/
__device__ inline static void reducedSpongeLyra(uint64_t *v) {
	int i;

	for (i = 0; i < RHO; i++){
		ROUND_LYRA(i);
	}
}

/**
* Performs the initial organization of parameters
* And starts the setup phase.
* Initializes the Sponge's State
* Sets the passwords + salt + params and makes the padding
* Absorb this data to the state.
* From setup:
* Initializes M[0]
* Initializes M[1]
* Initializes M[2]
*
* @param memMatrixGPU                  Matrix start
* @param pkeysGPU			The derived keys of each thread
* @param kLen				Desired key length
* @param pwdGPU			User password
* @param pwdlen			Password length
* @param saltGPU			Salt
* @param saltlen			Salt length
* @param timeCost                      Parameter to determine the processing time (T)
* @param nRows				Matrix total number of rows
* @param nCols				Matrix total number of columns
* @param nBlocksInput                  The number of blocks to be absorbed
* @param totalPasswords                Total number of passwords being tested
*/
__global__ void bootStrapGPU(uint64_t * memMatrixGPU, unsigned char * pkeysGPU, unsigned int kLen, unsigned char *pwdGPU, unsigned int pwdlen, unsigned char *saltGPU, unsigned int saltlen, unsigned int timeCost, unsigned int nRows, unsigned int nCols, uint64_t nBlocksInput, unsigned int totalPasswords) {
	int i;
	// Size of each chunk that each thread will work with
	//updates global sizeSlicedRows;
	sizeSlicedRows = (nRows / nPARALLEL) * ROW_LEN_INT64;
	byte *ptrByte;
	byte *ptrByteSource;
	int threadNumber;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		uint64_t sliceStart = threadNumber*sizeSlicedRows;
		uint64_t thStart = ((uint64_t)(threadNumber / nPARALLEL));

		//============= Padding (password + salt + params) with 10*1 ===============//
		//OBS.:The memory matrix will temporarily hold the password: not for saving memory,
		//but this ensures that the password copied locally will be overwritten as soon as possible
		ptrByte = (byte*)& memMatrixGPU[sliceStart];
		ptrByteSource = (byte*)& pwdGPU[thStart * pwdlen];

		//First, we clean enough blocks for the password, salt, params and padding
		for (i = 0; i < nBlocksInput * BLOCK_LEN_BLAKE2_SAFE_BYTES; i++) {
			ptrByte[i] = (byte)0;
		}

		//Prepends the password
		memcpy(ptrByte, ptrByteSource, pwdlen);
		ptrByte += pwdlen;

		//The indexed salt
		ptrByteSource = (byte*)& saltGPU[thStart * saltlen];

		//Concatenates the salt
		memcpy(ptrByte, ptrByteSource, saltlen);
		ptrByte += saltlen;

		//Concatenates the basil: every integer passed as parameter, in the order they are provided by the interface
		memcpy(ptrByte, &kLen, sizeof(int));
		ptrByte += sizeof(int);
		memcpy(ptrByte, &pwdlen, sizeof(int));
		ptrByte += sizeof(int);
		memcpy(ptrByte, &saltlen, sizeof(int));
		ptrByte += sizeof(int);
		memcpy(ptrByte, &timeCost, sizeof(int));
		ptrByte += sizeof(int);
		memcpy(ptrByte, &nRows, sizeof(int));
		ptrByte += sizeof(int);
		memcpy(ptrByte, &nCols, sizeof(int));
		ptrByte += sizeof(int);

#if (nPARALLEL > 1)
		//The difference from sequential version:
		//Concatenates the total number of threads
		int p = nPARALLEL;
		memcpy(ptrByte, &p, sizeof(int));
		ptrByte += sizeof(int);
		//Concatenates thread number
		int thread = threadNumber % nPARALLEL;
		memcpy(ptrByte, &thread, sizeof(int));

		ptrByte += sizeof(int);
#endif //nParallel > 1

		//Now comes the padding
		*ptrByte = 0x80; //first byte of padding: right after the password

		//resets the pointer to the start of the memory matrix
		ptrByte = (byte*)& memMatrixGPU[sliceStart];
		ptrByte += nBlocksInput * BLOCK_LEN_BLAKE2_SAFE_BYTES - 1; //sets the pointer to the correct position: end of incomplete block
		*ptrByte ^= 0x01; //last byte of padding: at the end of the last incomplete block
	}
}

/**
* Initializes the Sponge State. The first 512 bits are set to zeros and the remainder
* receive Blake2b's IV as per Blake2b's specification. <b>Note:</b> Even though sponges
* typically have their internal state initialized with zeros, Blake2b's G function
* has a fixed point: if the internal state and message are both filled with zeros. the
* resulting permutation will always be a block filled with zeros; this happens because
* Blake2b does not use the constants originally employed in Blake2 inside its G function,
* relying on the IV for avoiding possible fixed points.
*
* @param state             The 1024-bit array to be initialized
* @param totalPasswords    Total number of passwords being tested
*/
__global__ void initState(uint64_t state[/*16*/], unsigned int totalPasswords) {
	int threadNumber;
	uint64_t start;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		start = threadNumber * STATESIZE_INT64;
		//First 512 bis are zeros
		state[start + 0] = 0x0ULL;
		state[start + 1] = 0x0ULL;
		state[start + 2] = 0x0ULL;
		state[start + 3] = 0x0ULL;
		state[start + 4] = 0x0ULL;
		state[start + 5] = 0x0ULL;
		state[start + 6] = 0x0ULL;
		state[start + 7] = 0x0ULL;
		//Remainder BLOCK_LEN_BLAKE2_SAFE_BYTES are reserved to the IV
		state[start + 8] = blake2b_IV[0];
		state[start + 9] = blake2b_IV[1];
		state[start + 10] = blake2b_IV[2];
		state[start + 11] = blake2b_IV[3];
		state[start + 12] = blake2b_IV[4];
		state[start + 13] = blake2b_IV[5];
		state[start + 14] = blake2b_IV[6];
		state[start + 15] = blake2b_IV[7];
	}
}

/**
* Performs an absorb operation for a single block (BLOCK_LEN_BLAKE2_SAFE_INT64
* words of type uint64_t), using G function as the internal permutation
*
* @param state         The current state of the sponge
* @param in            The block to be absorbed (BLOCK_LEN_BLAKE2_SAFE_INT64 words)
*/
__device__ inline void absorbBlockBlake2Safe(uint64_t *state, const uint64_t *in) {
	//XORs the first BLOCK_LEN_BLAKE2_SAFE_INT64 words of "in" with the current state
	state[0] ^= in[0];
	state[1] ^= in[1];
	state[2] ^= in[2];
	state[3] ^= in[3];
	state[4] ^= in[4];
	state[5] ^= in[5];
	state[6] ^= in[6];
	state[7] ^= in[7];

	//Applies the transformation f to the sponge's state
	spongeLyra(state);
}

/**
* Performs a initial absorb operation
* Absorbs salt, password and the other parameters
*
* @param memMatrixGPU		Matrix start
* @param stateThreadGPU	The current state of the sponge
* @param stateIdxGPU  		Index of the threads, to be absorbed
* @param nBlocksInput 		The number of blocks to be absorbed
* @param totalPasswords        Total number of passwords being tested
*/
__global__ void absorbInput(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, uint64_t *stateIdxGPU, uint64_t nBlocksInput, unsigned int totalPasswords) {
	uint64_t *ptrWord;
	uint64_t *threadState;
	int threadNumber;
	uint64_t kP;
	uint64_t sliceStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		sliceStart = threadNumber*sizeSlicedRows;
		threadState = (uint64_t *)& stateThreadGPU[threadNumber * STATESIZE_INT64];

		//Absorbing salt, password and params: this is the only place in which the block length is hard-coded to 512 bits, for compatibility with Blake2b and BlaMka
		ptrWord = (uint64_t *)& memMatrixGPU[sliceStart];      //threadSliceMatrix;
		for (kP = 0; kP < nBlocksInput; kP++) {
			absorbBlockBlake2Safe(threadState, ptrWord);        //absorbs each block of pad(pwd || salt || params)
			ptrWord += BLOCK_LEN_BLAKE2_SAFE_INT64;             //BLOCK_LEN_BLAKE2_SAFE_INT64;  //goes to next block of pad(pwd || salt || params)
		}
	}
}

/**
* Performs a reduced squeeze operation for a single row, from the highest to
* the lowest index, using the reduced-round G function as the
* internal permutation
*
* @param state             The current state of the sponge
* @param rowOut            Row to receive the data squeezed
* @param totalPasswords    Total number of passwords being tested
*/
__global__ void reducedSqueezeRow0(uint64_t* rowOut, uint64_t* state, unsigned int totalPasswords) {
	int threadNumber;
	uint64_t sliceStart;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {
		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;

		uint64_t* ptrWord = &rowOut[sliceStart + (N_COLS - 1) * BLOCK_LEN_INT64]; //In Lyra2: pointer to M[0][C-1]
		int i, j;
		//M[0][C-1-col] = H.reduced_squeeze()
		for (i = 0; i < N_COLS; i++) {
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWord[j] = state[stateStart + j];
			}

			//Goes to next block (column) that will receive the squeezed data
			ptrWord -= BLOCK_LEN_INT64;

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);
		}
	}
}

/**
* Performs a reduced duplex operation for a single row, from the highest to
* the lowest index of its columns, using the reduced-round G function
* as the internal permutation
*
* @param state		        The current state of the sponge
* @param rowIn		        Matrix start (base row)
* @param first		        Index used with rowIn to calculate wich row will feed the sponge
* @param second	        Index used with rowIn to calculate wich row will be feeded with sponge state
* @param totalPasswords        Total number of passwords being tested
*/
__global__ void reducedDuplexRow1and2(uint64_t *rowIn, uint64_t *state, unsigned int totalPasswords, int first, int second) {
	int i, j;

	int threadNumber;
	uint64_t sliceStart;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;

		//Row to feed the sponge
		uint64_t* ptrWordIn = (uint64_t*)& rowIn[sliceStart + first * ROW_LEN_INT64]; //In Lyra2: pointer to prev
		//Row to receive the sponge's output
		uint64_t* ptrWordOut = (uint64_t*)& rowIn[sliceStart + second * ROW_LEN_INT64 + (N_COLS - 1) * BLOCK_LEN_INT64]; //In Lyra2: pointer to row

		for (i = 0; i < N_COLS; i++) {

			//Absorbing "M[0][col]"
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				state[stateStart + j] ^= (ptrWordIn[j]);
			}

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);

			//M[1][C-1-col] = M[0][col] XOR rand
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordOut[j] = ptrWordIn[j] ^ state[stateStart + j];
			}

			//Input: next column (i.e., next block in sequence)
			ptrWordIn += BLOCK_LEN_INT64;
			//Output: goes to previous column
			ptrWordOut -= BLOCK_LEN_INT64;
		}
	}
}

/**
* Performs a duplexing operation over
* "M[rowInOut0][col] [+] M[rowInOut1][col] [+] M[rowIn0][col_0] [+] M[rowIn1][col_1]",
* where [+] denotes wordwise addition, ignoring carries between words. The value of
* "col_0" is computed as "lsw(rot^2(rand)) mod N_COLS", and "col_1" as
* "lsw(rot^3(rand)) mod N_COLS", where lsw() means "the least significant word"
* where rot is a right rotation by 'omega' bits (e.g., 1 or more words)
* N_COLS is a system parameter, and "rand" corresponds
* to the sponge's output for each column absorbed.
* The same output is then employed to make
* "M[rowInOut0][col] = M[rowInOut0][col] XOR rand" and
* "M[rowInOut1][col] = M[rowInOut1][col] XOR rot(rand)".
*
* @param memMatrixGPU          Matrix start
* @param state                 The current state of the sponge
* @param prev0                 Another row used only as input
* @param prev1                 Stores the previous value of row1
* @param row0			        Row used as input and to receive output after rotation
* @param row1			        Pseudorandom indice to a row from another slice, used only as input
* @param totalPasswords        Total number of passwords being tested
*/
__device__ void reducedDuplexRowWandering_P1(uint64_t *memMatrixGPU, uint64_t *state, uint64_t prev0, uint64_t row0, uint64_t row1, uint64_t prev1, unsigned int totalPasswords) {
	int threadNumber;
	uint64_t sliceStart;
	uint64_t stateStart;
	uint64_t randomColumn0; //In Lyra2: col0
	uint64_t randomColumn1; //In Lyra2: col1

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;


		uint64_t* ptrWordInOut0 = (uint64_t *)& memMatrixGPU[sliceStart + (row0 * ROW_LEN_INT64)]; //In Lyra2: pointer to row0
		uint64_t* ptrWordInOut1 = (uint64_t *)& memMatrixGPU[sliceStart + (row1 * ROW_LEN_INT64)]; //In Lyra2: pointer to row0_p
		uint64_t* ptrWordIn0; //In Lyra2: pointer to prev0
		uint64_t* ptrWordIn1; //In Lyra2: pointer to prev1

		int i, j;

		for (i = 0; i < N_COLS; i++) {
			//col0 = lsw(rot^2(rand)) mod N_COLS
			//randomColumn0 = ((uint64_t)state[stateStart + 4] & (N_COLS-1))*BLOCK_LEN_INT64;           /*(USE THIS IF N_COLS IS A POWER OF 2)*/
			randomColumn0 = ((uint64_t)state[stateStart + 4] % N_COLS) * BLOCK_LEN_INT64;              /*(USE THIS FOR THE "GENERIC" CASE)*/
			ptrWordIn0 = (uint64_t *)& memMatrixGPU[sliceStart + (prev0 * ROW_LEN_INT64) + randomColumn0];

			//col0 = LSW(rot^3(rand)) mod N_COLS
			//randomColumn1 = ((uint64_t)state[stateStart + 6] & (N_COLS-1))*BLOCK_LEN_INT64;           /*(USE THIS IF N_COLS IS A POWER OF 2)*/
			randomColumn1 = ((uint64_t)state[stateStart + 6] % N_COLS) * BLOCK_LEN_INT64;              /*(USE THIS FOR THE "GENERIC" CASE)*/
			ptrWordIn1 = (uint64_t *)& memMatrixGPU[sliceStart + (prev1 * ROW_LEN_INT64) + randomColumn1];

			//Absorbing "M[row0] [+] M[row1] [+] M[prev0] [+] M[prev1]"
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				state[stateStart + j] ^= (ptrWordInOut0[j] + ptrWordInOut1[j] + ptrWordIn0[j] + ptrWordIn1[j]);
			}

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);

			//M[rowInOut0][col] = M[rowInOut0][col] XOR rand
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordInOut0[j] ^= state[stateStart + j];
			}

			//M[rowInOut1][col] = M[rowInOut1][col] XOR rot(rand)
			//rot(): right rotation by 'omega' bits (e.g., 1 or more words)
			//we rotate 2 words for compatibility with the SSE implementation
			for (j = 0; j < BLOCK_LEN_INT64; j++){
				ptrWordInOut1[j] ^= state[stateStart + ((j + 2) % BLOCK_LEN_INT64)];
			}

			//Goes to next block
			ptrWordInOut0 += BLOCK_LEN_INT64;
			ptrWordInOut1 += BLOCK_LEN_INT64;

		}
	}
}

/**
* Performs a duplexing operation over
* "M[rowInOut0][col] [+] M[rowInP][col] [+] M[rowIn0][col_0]",
* where [+] denotes wordwise addition, ignoring carries between words. The value of
* "col_0" is computed as "LSW(rot^3(rand)) mod N_COLS",where LSW means "the less significant word"
* (assuming 64-bit words), rot is a 128-bit  rotation to the right,
* N_COLS is a system parameter, and "rand" corresponds
* to the sponge's output for each column absorbed.
* The same output is then employed to make
* "M[rowInOut0][col] = M[rowInOut0][col] XOR rand".
*
* @param memMatrixGPU          Matrix start
* @param state                 The current state of the sponge
* @param prev0                 Another row used only as input
* @param row0			Row used as input and to receive output after rotation
* @param rowP			Pseudorandom indice to a row from another slice, used only as input
* @param window		Visitation window (equals a half slice)
* @param jP			Index to another slice of matrix
* @param totalPasswords        Total number of passwords being tested
*/
__device__ void reducedDuplexRowWanderingParallel(uint64_t *memMatrixGPU, uint64_t *state, uint64_t prev0, uint64_t row0, uint64_t rowP, uint64_t window, uint64_t jP, unsigned int totalPasswords) {
	int threadNumber;
	uint64_t sliceStart;
	uint64_t stateStart;
	uint64_t sliceStartjP;
	uint64_t randomColumn0; //In Lyra2: col0

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;

		//jP slice must be inside the  password´s thread pool
		//The integer part of threadNumber/nPARALLEL multiplied by nPARALLEL is the Base Slice Start for the password thread pool
		sliceStartjP = ((((uint64_t)(threadNumber / nPARALLEL)) * nPARALLEL) + jP) * sizeSlicedRows;

		//Row used as input and to receive output after rotation
		uint64_t* ptrWordInOut0 = (uint64_t *)& memMatrixGPU[sliceStart + (row0 * ROW_LEN_INT64)]; //In Lyra2: pointer to row0
		//Row used only as input
		uint64_t* ptrWordInP = (uint64_t *)& memMatrixGPU[sliceStartjP + (rowP * ROW_LEN_INT64)]; //In Lyra2: pointer to row0_p
		//Another row used only as input
		uint64_t* ptrWordIn0; //In Lyra2: pointer to prev0

		int i, j;

		for (i = 0; i < N_COLS; i++) {
			//col0 = LSW(rot^3(rand)) mod N_COLS
			//randomColumn0 = ((uint64_t)state[stateStart + 6] & (N_COLS-1))*BLOCK_LEN_INT64;           /*(USE THIS IF N_COLS IS A POWER OF 2)*/
			randomColumn0 = ((uint64_t)state[stateStart + 6] % N_COLS) * BLOCK_LEN_INT64;              /*(USE THIS FOR THE "GENERIC" CASE)*/

			ptrWordIn0 = (uint64_t *)& memMatrixGPU[sliceStart + (prev0 * ROW_LEN_INT64) + randomColumn0];

			//Absorbing "M[row0] [+] M[prev0] [+] M[rowP]"
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				state[stateStart + j] ^= (ptrWordInOut0[j] + ptrWordIn0[j] + ptrWordInP[j]);
			}

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);

			//M[rowInOut0][col] = M[rowInOut0][col] XOR rand
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordInOut0[j] ^= state[stateStart + j];
			}

			//Goes to next block
			ptrWordInOut0 += BLOCK_LEN_INT64;
			ptrWordInP += BLOCK_LEN_INT64;

		}
	}
}

/**
* Performs an absorb operation of single column from "in", the
* said column being pseudorandomly picked in the range [0, BLOCK_LEN_INT64[,
* using the full-round G function as the internal permutation
*
* @param state The current state of the sponge
* @param in    			Matrix start
* @param row0				The row whose column (BLOCK_LEN_INT64 words) should be absorbed
* @param randomColumn0                 The random column to be absorbed
* @param totalPasswords                Total number of passwords being tested
*/
__device__ void absorbRandomColumn(uint64_t *in, uint64_t *state, uint64_t row0, uint64_t randomColumn0, unsigned int totalPasswords) {
	int i;
	int threadNumber;
	uint64_t sliceStart;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;

		uint64_t* ptrWordIn = (uint64_t*)& in[sliceStart + (row0 * ROW_LEN_INT64) + randomColumn0];

		//absorbs the column picked
		for (i = 0; i < BLOCK_LEN_INT64; i++) {
			state[stateStart + i] ^= ptrWordIn[i];
		}

		//Applies the full-round transformation f to the sponge's state
		spongeLyra(&state[stateStart]);
	}
}

/**
* Wandering phase: performs the visitation loop
* Visitation loop chooses pseudo random rows (row0 and row1) based in state content
* And performs a reduced-round duplexing operation over:
* "M[row0][col] [+] M[row1][col] [+] M[prev0][col0] [+] M[prev1][col1]
* Updating both M[row0] and M[row1] using the output to make:
* M[row0][col] = M[row0][col] XOR rand;
* M[row1][col] = M[row1][col] XOR rot(rand)
* Where rot() is a right rotation by 'omega' bits (e.g., 1 or more words)
*
* @param stateThreadGPU      	The current state of the sponge
* @param memMatrixGPU          Array that will receive the data squeezed
* @param timeCost            	Parameter to determine the processing time (T)
* @param nRows         		Number of rows
* @param totalPasswords        Total number of passwords being tested
* @param prev0                 Stores the previous value of row0, the last row ever initialized
* @param prev1                 Stores the previous value of row1
*/
__device__ void wanderingPhaseGPU2_P1(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, unsigned int timeCost, uint64_t nRows, unsigned int totalPasswords, uint64_t prev0, uint64_t prev1) {
	uint64_t wCont;             //Time Loop iterator
	uint64_t row0;              //row0: sequentially written during Setup; randomly picked during Wandering
	uint64_t row1;              //rowP: revisited during Setup, and then read [and written]; randomly picked during Wandering
	uint64_t threadNumber;

	uint64_t stateStart;


	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;

		for (wCont = 0; wCont < timeCost * nRows; wCont++) {
			//Selects a pseudorandom indices row0 and rowP (row0 = LSW(rand) mod wnd and rowP = LSW(rot(rand)) mod wnd)
			//------------------------------------------------------------------------------------------
			//(USE THIS IF window IS A POWER OF 2)
			//row0 = (((uint64_t)stateThreadGPU[stateStart + 0]) & nRows);
			//row1 = (((uint64_t)stateThreadGPU[stateStart + 2]) & nRows);
			//(USE THIS FOR THE "GENERIC" CASE)
			row0 = (((uint64_t)stateThreadGPU[stateStart + 0]) % nRows);   //row0 = lsw(rand) mod nRows
			row1 = (((uint64_t)stateThreadGPU[stateStart + 2]) % nRows);   //row1 = lsw(rot(rand)) mod nRows
			//we rotate 2 words for compatibility with the SSE implementation

			//Performs a reduced-round duplexing operation over "M[row0][col] [+] M[row1][col] [+] M[prev0][col0] [+] M[prev1][col1], updating both M[row0] and M[row1]
			//M[row0][col] = M[row0][col] XOR rand;
			//M[row1][col] = M[row1][col] XOR rot(rand)                     rot(): right rotation by 'omega' bits (e.g., 1 or more words)
			reducedDuplexRowWandering_P1(memMatrixGPU, stateThreadGPU, prev0, row0, row1, prev1, totalPasswords);

			//update prev: they now point to the last rows ever updated
			prev0 = row0;
			prev1 = row1;

		}

		//============================ Wrap-up Phase ===============================//
		//Absorbs one last block of the memory matrix with the full-round sponge
		absorbRandomColumn(memMatrixGPU, stateThreadGPU, row0, 0, totalPasswords);
	}

}

/**
* Wandering phase: performs the visitation loop
* Visitation loop chooses pseudo random rows (row0 and rowP) based in state content
* And performs a reduced-round duplexing operation over:
* M[row0] [+] Mj[rowP] [+] M[prev0]
* Updating M[row0] using the output from reduced-round duplexing (rand):
* M[row0][col] = M[row0][col] XOR rand
*
* @param stateThreadGPU      	The current state of the sponge
* @param memMatrixGPU          Array that will receive the data squeezed
* @param timeCost        	Parameter to determine the processing time (T)
* @param sizeSlice		Number of rows for each thread
* @param totalPasswords        Total number of passwords being tested
* @param sqrt                  To control step changes in visitation
* @param prev0                 Stores the previous value of row0, the last row ever initialized
*/
__device__ void wanderingPhaseGPU2(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, unsigned int timeCost, uint64_t sizeSlice, unsigned int totalPasswords, uint64_t sqrt, uint64_t prev0) {
	uint64_t wCont;             //Time Loop iterator
	uint64_t window;            //Visitation window (used to define which rows can be revisited during Setup)
	uint64_t row0;              //row0: sequentially written during Setup; randomly picked during Wandering

	uint64_t rowP;              //rowP: revisited during Setup, and then read [and written]; randomly picked during Wandering
	uint64_t jP;                //Index to another thread
	uint64_t threadNumber;

	uint64_t stateStart;

	uint64_t off0;              //complementary offsets to calculate row0
	uint64_t offP;              //complementary offsets to calculate rowP
	uint64_t offTemp;

	uint64_t sync = sqrt;

	uint64_t halfSlice = sizeSlice / 2;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;

		window = halfSlice;
		off0 = 0;
		offP = window;

		for (wCont = 0; wCont < timeCost * sizeSlice; wCont++) {
			//Selects a pseudorandom indices row0 and rowP (row0 = LSW(rand) mod wnd and rowP = LSW(rot(rand)) mod wnd)
			//------------------------------------------------------------------------------------------
			//(USE THIS IF window IS A POWER OF 2)
			//row0  = off0 + (((uint64_t)stateThreadGPU[stateStart + 0]) & (window-1));
			//row0P = offP + (((uint64_t)stateThreadGPU[stateStart + 2]) & (window-1));
			//(USE THIS FOR THE "GENERIC" CASE)
			row0 = off0 + (((uint64_t)stateThreadGPU[stateStart + 0]) % window);
			rowP = offP + (((uint64_t)stateThreadGPU[stateStart + 2]) % window);

			//Selects a pseudorandom indices j0 (LSW(rot^2 (rand)) mod p)
			jP = ((uint64_t)stateThreadGPU[stateStart + 4]) % nPARALLEL;

			//Performs a reduced-round duplexing operation over M[row0] [+] Mj[rowP] [+] M[prev0], updating M[row0]
			//M[row0][col] = M[row0][col] XOR rand;
			reducedDuplexRowWanderingParallel(memMatrixGPU, stateThreadGPU, prev0, row0, rowP, window, jP, totalPasswords);

			//update prev: they now point to the last rows ever updated
			prev0 = row0;

			if (wCont == sync) {
				sync += sqrt;
				offTemp = off0;
				off0 = offP;
				offP = offTemp;
				__syncthreads();
			}
		}
		__syncthreads();

		//============================ Wrap-up Phase ===============================//
		//Absorbs one last block of the memory matrix with the full-round sponge
		absorbRandomColumn(memMatrixGPU, stateThreadGPU, row0, 0, totalPasswords);
	}

}

/**
* Performs a duplexing operation over
* "M[rowInOut][col] [+] M[rowIn0][col] [+] M[rowIn1][col]", where [+] denotes
* wordwise addition, ignoring carries between words, , for all values of "col"
* in the [0,N_COLS[ interval. The  output of this operation, "rand", is then
* employed to make
* "M[rowOut][(N_COLS-1)-col] = M[rowIn0][col] XOR rand" and
* "M[rowInOut][col] =  M[rowInOut][col] XOR rot(rand)",
* where rot is a right rotation by 'omega' bits (e.g., 1 or more words)
* and N_COLS is a system parameter.
*
* @param state                         The current state of the sponge
* @param memMatrixGPU                  Matrix start
* @param prev0                         Index to calculate rowIn0, the previous row0
* @param prev1                         Index to calculate rowIn1
* @param row0                          Index to calculate rowOut, the row being initialized
* @param row1                          Index to calculate rowInOut, the row to be revisited and updated
* @param totalPasswords                Total number of passwords being tested
*/
__device__ void reducedDuplexRowFilling_P1(uint64_t *state, uint64_t *memMatrixGPU, uint64_t prev0, uint64_t prev1, uint64_t row0, uint64_t row1, unsigned int totalPasswords) {
	int i, j;
	int threadNumber;

	uint64_t sliceStart;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {
		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;     //sizeSlicedRows = (nRows/nPARALLEL) * ROW_LEN_INT64

		//Row used only as input (rowIn0 or M[prev0])
		uint64_t* ptrWordIn0 = (uint64_t *)& memMatrixGPU[sliceStart + prev0 * ROW_LEN_INT64];         //In Lyra2: pointer to prev0, the last row ever initialized

		//Another row used only as input (rowIn1 or M[prev1])
		uint64_t* ptrWordIn1 = (uint64_t *)& memMatrixGPU[sliceStart + prev1 * ROW_LEN_INT64];     //In Lyra2: pointer to prev1, the last row ever revisited and updated

		//Row used as input and to receive output after rotation (rowInOut or M[row1])
		uint64_t* ptrWordInOut = (uint64_t *)& memMatrixGPU[sliceStart + row1 * ROW_LEN_INT64];    //In Lyra2: pointer to row1, to be revisited and updated

		//Row receiving the output (rowOut or M[row0])
		uint64_t* ptrWordOut = (uint64_t *)& memMatrixGPU[sliceStart + (row0 * ROW_LEN_INT64) + ((N_COLS - 1) * BLOCK_LEN_INT64)]; //In Lyra2: pointer to row0, to be initialized

		for (i = 0; i < N_COLS; i++) {
			//Absorbing "M[row1] [+] M[prev0] [+] M[prev1]"
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				state[stateStart + j] ^= (ptrWordInOut[j] + ptrWordIn0[j] + ptrWordIn1[j]);
			}

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);

			//M[row0][col] = M[prev0][col] XOR rand
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordOut[j] = ptrWordIn0[j] ^ state[stateStart + j];
			}

			//M[row1][col] = M[row1][col] XOR rot(rand)
			//rot(): right rotation by 'omega' bits (e.g., 1 or more words)
			//we rotate 2 words for compatibility with the SSE implementation
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordInOut[j] ^= state[stateStart + ((j + 2) % BLOCK_LEN_INT64)]; // BLOCK_LEN_INT64 = 12
			}

			//Inputs: next column (i.e., next block in sequence)
			ptrWordInOut += BLOCK_LEN_INT64;
			ptrWordIn0 += BLOCK_LEN_INT64;
			ptrWordIn1 += BLOCK_LEN_INT64;
			//Output: goes to previous column
			ptrWordOut -= BLOCK_LEN_INT64;
		}
	}
}



/**
* Performs a duplexing operation over
* "M[rowInOut][col] [+] M[rowIn0][col] [+] M[rowIn1][col]", where [+] denotes
* wordwise addition, ignoring carries between words, , for all values of "col"
* in the [0,N_COLS[ interval. The  output of this operation, "rand", is then
* employed to make
* "M[rowOut][(N_COLS-1)-col] = M[rowIn0][col] XOR rand" and
* "M[rowInOut][col] =  M[rowInOut][col] XOR rot(rand)",
* where rot is a right rotation by 'omega' bits (e.g., 1 or more words)
* and N_COLS is a system parameter.
*
* @param state                         The current state of the sponge
* @param memMatrixGPU                  Matrix start
* @param prev0                         Index to calculate rowIn0, the previous row0
* @param prevP                         Index to calculate rowIn1
* @param row0                          Index to calculate rowOut, the row being initialized
* @param rowP                          Index to calculate rowInOut, the row to be revisited and updated
* @param jP                            Index to another slice of matrix (slice belonging to another thread)
* @param totalPasswords                Total number of passwords being tested
*/
__device__ void reducedDuplexRowFilling(uint64_t *state, uint64_t *memMatrixGPU, uint64_t prev0, uint64_t prevP, uint64_t row0, uint64_t rowP, uint64_t jP, unsigned int totalPasswords) {
	int i, j;
	int threadNumber;

	uint64_t sliceStart;
	uint64_t sliceStartjP;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {
		stateStart = threadNumber * STATESIZE_INT64;
		sliceStart = threadNumber * sizeSlicedRows;     //sizeSlicedRows = (nRows/nPARALLEL) * ROW_LEN_INT64
		//jP slice must be inside the  password´s thread pool
		//The integer part of threadNumber/nPARALLEL multiplied by nPARALLEL is the Base Slice Start for the password thread pool
		sliceStartjP = ((((uint64_t)(threadNumber / nPARALLEL)) * nPARALLEL) + jP) * sizeSlicedRows;

		//Row used only as input
		uint64_t* ptrWordIn0 = (uint64_t *)& memMatrixGPU[sliceStart + prev0 * ROW_LEN_INT64];         //In Lyra2: pointer to prev0, the last row ever initialized

		//Another row used only as input
		uint64_t* ptrWordIn1 = (uint64_t *)& memMatrixGPU[sliceStartjP + (prevP * ROW_LEN_INT64)];     //In Lyra2: pointer to prev1, the last row ever revisited and updated

		//Row used as input and to receive output after rotation
		uint64_t* ptrWordInOut = (uint64_t *)& memMatrixGPU[sliceStartjP + (rowP * ROW_LEN_INT64)];    //In Lyra2: pointer to row1, to be revisited and updated

		//Row receiving the output
		uint64_t* ptrWordOut = (uint64_t *)& memMatrixGPU[sliceStart + (row0 * ROW_LEN_INT64) + ((N_COLS - 1) * BLOCK_LEN_INT64)]; //In Lyra2: pointer to row0, to be initialized

		for (i = 0; i < N_COLS; i++) {
			//Absorbing "M[rowP] [+] M[prev0] [+] M[prev1]"
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				state[stateStart + j] ^= (ptrWordInOut[j] + ptrWordIn0[j] + ptrWordIn1[j]);
			}

			//Applies the reduced-round transformation f to the sponge's state
			reducedSpongeLyra(&state[stateStart]);

			//M[row0][col] = M[prev0][col] XOR rand
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordOut[j] = ptrWordIn0[j] ^ state[stateStart + j];
			}

			//M[rowP][col] = M[rowP][col] XOR rot(rand)
			//rot(): right rotation by 'omega' bits (e.g., 1 or more words)
			//we rotate 2 words for compatibility with the SSE implementation
			for (j = 0; j < BLOCK_LEN_INT64; j++) {
				ptrWordInOut[j] ^= state[stateStart + ((j + 2) % BLOCK_LEN_INT64)]; // BLOCK_LEN_INT64 = 12
			}

			//Inputs: next column (i.e., next block in sequence)
			ptrWordInOut += BLOCK_LEN_INT64;
			ptrWordIn0 += BLOCK_LEN_INT64;
			ptrWordIn1 += BLOCK_LEN_INT64;
			//Output: goes to previous column
			ptrWordOut -= BLOCK_LEN_INT64;
		}
	}
}

/**
* Performs matrix initialization and calls wandering phase
* During setup, performs a reduced-round duplexing operation over:
* "Mj[rowP][col] [+] Mi[prev0][col] [+] Mj[prevP][col]", filling Mi[row0] and updating Mj[rowP]
* M[row0][N_COLS-1-col] = M[prev0][col] XOR rand;
* Mj[rowP][col] = Mj[rowP][col] XOR rot(rand)
* Where rot() is a right rotation by 'omega' bits (e.g., 1 or more words)
* and N_COLS is a system parameter.
*
* @param memMatrixGPU		Matrix start
* @param stateThreadGPU	The current state of the sponge
* @param sizeSlice		Number of rows for each thread
* @param totalPasswords        Total number of passwords being tested
* @param timeCost        	Parameter to determine the processing time (T)
*/
__global__ void setupPhaseWanderingGPU(uint64_t * memMatrixGPU, uint64_t * stateThreadGPU, uint64_t sizeSlice, unsigned int totalPasswords, unsigned int timeCost) {
	uint64_t step = 1;          //Visitation step (used during Setup and Wandering phases)
	uint64_t window = 2;        //Visitation window (used to define which rows can be revisited during Setup)
	int64_t gap = 1;            //Modifier to the step, assuming the values 1 or -1

	uint64_t row0 = 3;          //row0: sequentially written during Setup; randomly picked during Wandering
	uint64_t prev0 = 2;         //prev0: stores the previous value of row0
	uint64_t rowP = 1;          //rowP: revisited during Setup, and then read [and written]; randomly picked during Wandering
	uint64_t prevP = 0;         //prevP: stores the previous value of rowP
	uint64_t jP;                //Index to another thread, starts with threadNumber
	uint64_t sync = 4;          //Synchronize counter
	uint64_t sqrt = 2;          //Square of window (i.e., square(window)), when a window is a square number;
	//otherwise, sqrt = 2*square(window/2)

	int threadNumber;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		//jP must be in the thread pool of the same password
		jP = threadNumber % nPARALLEL;

		//Filling Loop
		for (row0 = 3; row0 < sizeSlice; row0++) {
			//Performs a reduced-round duplexing operation over "Mj[rowP][col] [+] Mi[prev0][col] [+] Mj[prevP][col]", filling Mi[row0] and updating Mj[rowP]
			//Mi[row0][N_COLS-1-col] = Mi[prev0][col] XOR rand;
			//Mj[rowP][col] = Mj[rowP][col] XOR rot(rand)                    rot(): right rotation by 'omega' bits (e.g., 1 or more words)
			reducedDuplexRowFilling(stateThreadGPU, memMatrixGPU, prev0, prevP, row0, rowP, jP, totalPasswords);

			//Updates the "prev" indices: the rows more recently updated
			prev0 = row0;
			prevP = rowP;

			//updates the value of row1: deterministically picked, with a variable step
			rowP = (rowP + step) & (window - 1);

			//Checks if all rows in the window where visited.
			if (rowP == 0) {
				window *= 2;            //doubles the size of the re-visitation window
				step = sqrt + gap;      //changes the step
				gap = -gap;             //inverts the modifier to the step
				if (gap == -1) {
					sqrt *= 2;          //Doubles sqrt every other iteration
				}
			}
			if (row0 == sync) {
				sync += sqrt / 2;               //increment synchronize counter
				jP = (jP + 1) % nPARALLEL;      //change the visitation thread
				__syncthreads();
			}
		}

		//Waits all threads
		__syncthreads();

		//Now goes to Wandering Phase and the Absorb from Wrap-up
		//============================ Wandering Phase =============================//
		//=====Iteratively overwrites pseudorandom cells of the memory matrix=======//
		wanderingPhaseGPU2(memMatrixGPU, stateThreadGPU, timeCost, sizeSlice, totalPasswords, sqrt, prev0);

	}
}

/**
* Performs a squeeze operation, using G function as the
* internal permutation
*
* @param state             The current state of the sponge
* @param out               Array that will receive the data squeezed
* @param len               The number of bytes to be squeezed into the "out" array
* @param totalPasswords    Total number of passwords being tested
*/
__global__ void squeeze(uint64_t *state, byte *out, unsigned int len, unsigned int totalPasswords) {
	int i;
	int fullBlocks = len / BLOCK_LEN_BYTES;

	int threadNumber;
	uint64_t stateStart;

	// Thread index:
	threadNumber = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (threadNumber < (nPARALLEL * totalPasswords)) {

		stateStart = threadNumber * STATESIZE_INT64;

		byte *ptr = (byte *)& out[threadNumber * len];

		//Squeezes full blocks
		for (i = 0; i < fullBlocks; i++) {
			memcpy(ptr, &state[stateStart], BLOCK_LEN_BYTES);
			spongeLyra(&state[stateStart]);
			ptr += BLOCK_LEN_BYTES;
		}

		//Squeezes remaining bytes
		memcpy(ptr, &state[stateStart], (len % BLOCK_LEN_BYTES));
	}
}


/**
* Generates the passwords for Lyra2 attack.
*
* @param t_cost            Parameter to determine the processing time (T)
* @param m_cost            Memory cost parameter (defines the number of rows of the memory matrix, R)
* @param totalPasswords    Total number of passwords being tested
* @param gridSize          GPU grid configuration
* @param blockSize         GPU block configuration
* @param printKeys         Defines if the resulting keys will be in the output
*/
void multPasswordCUDA(unsigned int t_cost, unsigned int m_cost, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize, unsigned int printKeys) {
	//=================== Basic variables, with default values =======================//
	int kLen = 32;
	unsigned char *ptrChar;
	int pwdLen = 10;
	int saltLen = 10;
	int i, j;
	int result;
	//==========================================================================/

	if (m_cost / nPARALLEL < 4) {
		printf("Number of rows too small\n");
		exit(0);
	}

	size_t sizeMemMatrix = (size_t)((size_t)m_cost * (size_t)ROW_LEN_BYTES);

	printf("Total time cost: %d \n", t_cost);
	printf("Total number of rows: %d \n", m_cost);
	printf("Total number of cols: %d \n", N_COLS);
	char *spongeName = "";
	spongeName = "Blake2";
	printf("Sponge: %s\n", spongeName);
	printf("Total number of password: %d \n", totalPasswords);
	printf("Password length: %d \n", pwdLen);
	printf("Parallelism inside password derivation: %d \n", nPARALLEL);
	printf("Grid Size (blocks): %d\n", gridSize);
	printf("Block Size (threads): %d\n", blockSize);
	printf("BlockSize x GridSize (threads): %d\n", gridSize*blockSize);
	printf("Total number of threads: %d \n", nPARALLEL*totalPasswords);
	printf("Memory per password: %ld bytes (%ld MB)\n", (long int)sizeMemMatrix, (long int)(sizeMemMatrix) / (1024 * 1024));
	printf("Total Memory: %ld bytes (%ld MB)\n", (long int)sizeMemMatrix * totalPasswords, (long int)(sizeMemMatrix * totalPasswords) / (1024 * 1024));
	fflush(stdout);

	// All Keys:
	unsigned char *K = (unsigned char *)malloc(totalPasswords * kLen * sizeof(unsigned char));

	//Pointer to each passwords in the Matrix:
	unsigned char **passwords = (unsigned char **)malloc(totalPasswords * sizeof(unsigned char *));
	if (passwords == NULL) {
		printf("Memory allocation error in file: %s and line: %d\n", __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Matrix with all passwords:
	unsigned char *passwdMatrix = (unsigned char *)malloc(totalPasswords * pwdLen * sizeof(unsigned char));
	if (passwdMatrix == NULL) {
		printf("Memory allocation error in file: %s and line: %d\n", __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Pointer to each salt in the Matrix:
	unsigned char **salts = (unsigned char **)malloc(totalPasswords * sizeof(unsigned char *));
	if (salts == NULL) {
		printf("Memory allocation error in file: %s and line: %d\n", __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Matrix with all salts:
	unsigned char *saltMatrix = (unsigned char *)malloc(totalPasswords * saltLen * sizeof(unsigned char));
	if (saltMatrix == NULL) {
		printf("Memory allocation error in file: %s and line: %d\n", __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Places the pointers in the correct positions
	ptrChar = passwdMatrix;
	for (i = 0; i < totalPasswords; i++) {
		passwords[i] = ptrChar;
		ptrChar += pwdLen; // pwdLen * sizeof (unsigned char);
	}

	//Places the pointers in the correct positions
	ptrChar = saltMatrix;
	for (i = 0; i < totalPasswords; i++) {
		salts[i] = ptrChar;
		ptrChar += saltLen; // pwdLen * sizeof (unsigned char);
	}


#ifndef SAMEPASSWORD
#define SAMEPASSWORD 0
#endif
	//fills passwords
	for (i = 0; i < totalPasswords; i++) {
		for (j = 0; j < pwdLen; j++) {
#if SAMEPASSWORD == 1
			//Same password:
			passwords[i][j] = (0x30 + j);
#else
			//Different passwords:
			passwords[i][j] = (j + i*pwdLen) % 255;
#endif
		}
	}
	// fills salts
	for (i = 0; i < totalPasswords; i++) {
		for (j = 0; j < saltLen; j++) {
			salts[i][j] = (0x30 + j);
		}
	}

	/*
	printf("Number of Passwords: %d\n", totalPasswords);
	//Prints passwords
	printf("Passwords:\n");
	for (i = 0; i < totalPasswords; i++) {
	for (j = 0; j < pwdLen; j++) {
	printf("%2x|", passwords[i][j]);
	}
	printf("\n");
	}

	//Prints salts
	printf("Salts:\n");
	for (i = 0; i < totalPasswords; i++) {
	for (j = 0; j < saltLen; j++) {
	printf("%x|", salts[i][j]);
	}
	printf("\n");
	}
	*/

	struct timeval start;
	struct timeval end;
	gettimeofday(&start, NULL);

	//Calls the interface to the GPU program
	result = gpuMult(K, kLen, passwords, pwdLen, salts, saltLen, t_cost, m_cost, N_COLS, totalPasswords, gridSize, blockSize);

	gettimeofday(&end, NULL);
	unsigned long elapsed = (end.tv_sec - start.tv_sec) * 1000000 + end.tv_usec - start.tv_usec;

	if (result >= 0){
		//Prints returned keys
		if (printKeys == 1) {
			printf("Result of %d Keys:\n", totalPasswords);
			for (i = 0; i < totalPasswords; i++) {
				printf("Key #: %3d: ", i);
				for (j = 0; j < kLen; j++) {
					printf("%2x|", K[i*kLen + j]);
				}
				printf("\n");
			}
		}
	}

	if (result < 0) {
		printf("Execution Error!!!\n");
	}
	else {
		printf("Execution Time: %lu us (%.3f ms, %.3f seg)\n", elapsed, (float)elapsed / 1000, (float)elapsed / (1000 * 1000));
		printf("Execution Time per password: %.3f us (%.3f ms, %.3f seg)\n", (float)((float)elapsed / totalPasswords), (float)(((float)elapsed / totalPasswords) / 1000), (float)(((float)elapsed / totalPasswords) / (1000 * 1000)));
	}
	printf("------------------------------------------------------------------------------------------------------------------------------------------\n");

	hipDeviceReset();
	free(passwords);
	free(passwdMatrix);
	free(saltMatrix);
	free(salts);
	free(K);
}


int main(int argc, char *argv[]) {

	//=================== Basic variables, with default values =======================//
	unsigned int t_cost = 0;
	unsigned int m_cost = 0;
	unsigned int gridSize;
	unsigned int blockSize;
	unsigned int numberPasswds;
	//==========================================================================/

	//	Defines in which GPU will execute
	hipSetDevice(0);

	switch (argc) {
	case 2:
		if (strcmp(argv[1], "--help") == 0) {
			printf("Usage: \n");
			printf("%s tCost nRows --multPasswordCUDA totalPasswordsToTest totalBlocksToUse threadsPerBlock [optional print hash] (to test multiple GPU derivations in parallel)\n\n", argv[0]);
			return 0;
		}
		else {
			printf("Invalid options.\nFor more information, try \"%s --help\".\n", argv[0]);
			return 0;
		}

	case 7:
		if (strcmp(argv[3], "--multPasswordCUDA") == 0) {
			t_cost = atoi(argv[1]);
			m_cost = atoi(argv[2]);
			numberPasswds = atoi(argv[4]);
			gridSize = atoi(argv[5]);
			blockSize = atoi(argv[6]);
			multPasswordCUDA(t_cost, m_cost, numberPasswds, gridSize, blockSize, 0);
			return 0;
		}
		break;

	case 8:
		if (strcmp(argv[3], "--multPasswordCUDA") == 0) {
			t_cost = atoi(argv[1]);
			m_cost = atoi(argv[2]);
			numberPasswds = atoi(argv[4]);
			gridSize = atoi(argv[5]);
			blockSize = atoi(argv[6]);
			multPasswordCUDA(t_cost, m_cost, numberPasswds, gridSize, blockSize, 1);
			return 0;
		}
		break;
	default:
		printf("Invalid options.\nTry \"%s --help\" for help.\n", argv[0]);
		return 0;
	}
}
